#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
// #include <>

#include "CudaTests.h"

// #include <opencv4/opencv2/opencv.hpp>

// cv::Mat load_image(const char* image_path) {
//   cv::Mat image = cv::imread(image_path);
//   image.convertTo(image, CV_32FC3);
//   cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
//   return image;
// }

// void save_image(const char* output_filename,
//                 float* buffer,
//                 int height,
//                 int width) {
//   cv::Mat output_image(height, width, CV_32FC3, buffer);
//   // Make negative values zero.
//   cv::threshold(output_image,
//                 output_image,
//                 /*threshold=*/0,
//                 /*maxval=*/0,
//                 cv::THRESH_TOZERO);
//   cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
//   output_image.convertTo(output_image, CV_8UC3);
//   cv::imwrite(output_filename, output_image);
// }

// void load_image(std::string path)
// {
// 	ILuint imageName;
// 	ilGenImages(1, &imageName);
// 	ilBindImage(imageName);
// 	ilLoadImage(path.c_str());
// }

void test_cublas_vector_add()
{
    int n = 1 << 5;
    size_t bytes = n * sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b;

    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_c = (float *)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);

    for (int i = 0; i < n;i++)
    {
        h_a[i] = i+1;
        h_b[i] = 2 * i+1;
    }

    // hipblasHandle_t handle;
    // hipblasCreate(&handle);

    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

    const float scale = 2.0f;
    hipblasSaxpy(gCublasHandle, n, &scale, d_a, 1, d_b, 1);
    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

    for (int i = 0; i < n;i++)
    {
        printf("%f\n", h_c[i]);
    }

    // hipblasDestroy(gCublasHandle);

    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);
}

// Verify our result on the CPU
// Indexing must account for the CUBLAS operating on column-major data
void verify_solution(float *a, float *b, float *c, int M, int N, int K) {
  // Tolerance for our result (floats are imperfect)
  float epsilon = 0.001f;

  // For every row...
  for (int row = 0; row < M; row++) {
    // For every column
    for (int col = 0; col < N; col++) {
      // For every element in the row-col pair...
      float temp = 0;
      for (int i = 0; i < K; i++) {
        temp += a[row + M * i] * b[col * K + i];
      }

      // Check to see if the difference falls within our tolerance
      assert(fabs(c[col * M + row] - temp) <= epsilon);
    }
  }
}

void test_cublas_matmul()
{
  // Dimensions for our matrices
  // MxK * KxN = MxN
  const int M = 1 << 9;
  const int N = 1 << 8;
  const int K = 1 << 7;

  // Pre-calculate the size (in bytes) of our matrices
  const size_t bytes_a = M * K * sizeof(float);
  const size_t bytes_b = K * N * sizeof(float);
  const size_t bytes_c = M * N * sizeof(float);

  // Vectors for the host data
  std::vector<float> h_a(M * K);
  std::vector<float> h_b(K * N);
  std::vector<float> h_c(M * N);
  
  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // Pseudo random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set the seed
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  // Fill the matrix with random numbers on the device
  hiprandGenerateUniform(prng, d_a, M * K);
  hiprandGenerateUniform(prng, d_b, K * M);

  // cuBLAS handle
  // hipblasHandle_t handle;
  // hipblasCreate(&handle);

  // Scalaing factors
  float alpha = 1.0f;
  float beta = 0.0f;

  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb,
  // beta, C, ldc
  hipblasSgemm(gCublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, M, d_b, K,
              &beta, d_c, M);

  // Copy back the three matrices
  hipMemcpy(h_a.data(), d_a, bytes_a, hipMemcpyDeviceToHost);
  hipMemcpy(h_b.data(), d_b, bytes_b, hipMemcpyDeviceToHost);
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);

  // Verify solution
  verify_solution(h_a.data(), h_b.data(), h_c.data(), M, N, K);
  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free our memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

void test_cugemm()
{
	Tensor t1(make_shape(2, 3));
	t1(0, 0) = -1;
	t1(0, 1) = 1;
	t1(0, 2) = 4;
	t1(1, 0) = -4;
	t1(1, 1) = 0;
	t1(1, 2) = -3;
	t1.print();

	Tensor t1_t(make_shape(3, 2));
	t1_t(0, 0) = -1;
	t1_t(1, 0) = 1;
	t1_t(2, 0) = 4;
	t1_t(0, 1) = -4;
	t1_t(1, 1) = 0;
	t1_t(2, 1) = -3;
	t1_t.print();

	Tensor t2(make_shape(4, 2));
	t2(0, 0) = 2;
	t2(1, 0) = 3;
	t2(2, 0) = -2;
	t2(3, 0) = 1;
	t2(0, 1) = 4;
	t2(1, 1) = 0;
	t2(2, 1) = 5;
	t2(3, 1) = 6;
	t2.print();

	Tensor t2_t(make_shape(2, 4));
	t2_t(0, 0) = 2;
	t2_t(0, 1) = 3;
	t2_t(0, 2) = -2;
	t2_t(0, 3) = 1;
	t2_t(1, 0) = 4;
	t2_t(1, 1) = 0;
	t2_t(1, 2) = 5;
	t2_t(1, 3) = 6;
	t2_t.print();

	t1.copyToGPU();
	t1_t.copyToGPU();
	t2.copyToGPU();
	t2_t.copyToGPU();

	// printf("GPU print t1\n");
	// printGPU<<<1, 1>>>(t1.mShape[0], t1.mShape[1], t1.mLD, t1.mDataGPU);

	Tensor t3(make_shape(4, 3));

	//Mat Mul
	/*clblasDgemm(clblasRowMajor, clblasNoTrans, clblasNoTrans, t1.cols(), t2.rows(),
	t1.rows(), 1, t1.mData, t1.rows(), t2.mData, t2.rows(), 0, t3.mData, t3.rows())*/
	//cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, t1.rows(), t2.cols(),
	//	t1.cols(), 1, t1.mData, t1.cols(), t2.mData, t2.cols(), 0, t3.mData, t3.cols());
	gemm_gpu(&t1, &t2, &t3, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 0);
	// hipDeviceSynchronize();
	printf("1\n");
	// printGPU<<<1, 1>>>(t3.mShape[0],t3.mShape[1],t3.mLD,t3.mDataGPU);
	// hipDeviceSynchronize();
	t3.copyToCPU();
	t3.print();
	gemm_gpu(&t1_t, &t2_t, &t3, HIPBLAS_OP_T, HIPBLAS_OP_T, 1, 0);
	t3.copyToCPU();
	printf("2\n");
	t3.print();
	gemm_gpu(&t1, &t2_t, &t3, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, 0);
	t3.copyToCPU();
	printf("3\n");
	t3.print();
	gemm_gpu(&t1_t, &t2, &t3, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, 0);
	t3.copyToCPU();
	printf("4\n");
	t3.print();
	
	// Result should be:
	//  30  29  43  45
	// -29 -36 -19 -34

	printf("freeing mem\n");
	t1.freemem();
	t1_t.freemem();
	t2.freemem();
	t2_t.freemem();
	t3.freemem();
}

void test_cugemm_symm()
{
	Tensor t1(make_shape(2, 2));
	t1(0, 0) = -1;
	t1(0, 1) = 1;
	t1(1, 0) = -4;
	t1(1, 1) = 0;
	t1.print();

	Tensor t2(make_shape(2, 2));
	t2(0, 0) = 2;
	t2(0, 1) = 3;
	t2(1, 0) = 4;
	t2(1, 1) = 0;
	t2.print();

	t1.copyToGPU();
	// t1_t.copyToGPU();
	t2.copyToGPU();
	// t2_t.copyToGPU();

	printf("GPU print t1\n");
	printGPU<<<1, 1>>>(t1.mShape[0], t1.mShape[1], t1.mLD, t1.mDataGPU);

	Tensor t3(make_shape(2, 2));

	//Mat Mul
	/*clblasDgemm(clblasRowMajor, clblasNoTrans, clblasNoTrans, t1.cols(), t2.rows(),
	t1.rows(), 1, t1.mData, t1.rows(), t2.mData, t2.rows(), 0, t3.mData, t3.rows())*/
	//cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, t1.rows(), t2.cols(),
	//	t1.cols(), 1, t1.mData, t1.cols(), t2.mData, t2.cols(), 0, t3.mData, t3.cols());
	gemm_gpu(&t1, &t2, &t3, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 0);
	// hipDeviceSynchronize();
	printf("1\n");
	// printGPU<<<1, 1>>>(t3.mShape[0],t3.mShape[1],t3.mLD,t3.mDataGPU);
	// hipDeviceSynchronize();
	t3.copyToCPU();
	t3.print();
	// gemm_gpu(&t1_t, &t2_t, &t3, HIPBLAS_OP_T, HIPBLAS_OP_T, 1, 0);
	// t3.copyToCPU();
	// printf("2\n");
	// t3.print();
	// gemm_gpu(&t1, &t2_t, &t3, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, 0);
	// t3.copyToCPU();
	// printf("3\n");
	// t3.print();
	// gemm_gpu(&t1_t, &t2, &t3, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, 0);
	// t3.copyToCPU();
	// printf("4\n");
	// t3.print();
	
	// Result should be:
	//  30  29  43  45
	// -29 -36 -19 -34

	printf("freeing mem\n");
	t1.freemem();
	// t1_t.freemem();
	t2.freemem();
	// t2_t.freemem();
	t3.freemem();
}

void test_cudnn_forward()
{
	ilInit();
	std::string path = "tensorflow.png";
	// load_image("tensorflow.png");
	ILuint imageName;
	ilGenImages(1, &imageName);
	ilBindImage(imageName);
	ilLoadImage(path.c_str());
	// auto view = boost::gil::view(img);

	// auto channels = view.num_channels();
	// auto dim = view.dimensions();
	// printf("dim %d %d %d %d\n", dim.x, dim.y, dim.num_dimensions, channels);

	int ip_width = ilGetInteger(IL_IMAGE_WIDTH);
	int ip_height = ilGetInteger(IL_IMAGE_HEIGHT);
	int ip_channels = ilGetInteger(IL_IMAGE_CHANNELS);

	printf("dim %d %d %d\n", ip_width, ip_height, ip_channels);

	hipdnnTensorDescriptor_t input_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
											/*format=*/HIPDNN_TENSOR_NHWC,
											/*dataType=*/HIPDNN_DATA_FLOAT,
											/*batch_size=*/1,
											/*channels=*/4,
											/*image_height=*/ip_height,
											/*image_width=*/ip_width));

	hipdnnFilterDescriptor_t kernel_descriptor;
	checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
	checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
											/*dataType=*/HIPDNN_DATA_FLOAT,
											/*format=*/HIPDNN_TENSOR_NCHW,
											/*out_channels=*/4,
											/*in_channels=*/4,
											/*kernel_height=*/3,
											/*kernel_width=*/3));

	hipdnnConvolutionDescriptor_t convolution_descriptor;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
												/*pad_height=*/1,
												/*pad_width=*/1,
												/*vertical_stride=*/1,
												/*horizontal_stride=*/1,
												/*dilation_height=*/1,
												/*dilation_width=*/1,
												/*mode=*/HIPDNN_CROSS_CORRELATION,
												/*computeType=*/HIPDNN_DATA_FLOAT));

	int batch_size{0}, channels{0}, height{0}, width{0};
	checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
													input_descriptor,
													kernel_descriptor,
													&batch_size,
													&channels,
													&height,
													&width));

	std::cerr << "Output Image: " << batch_size << " batches of " << height << " x " << width << " x " << channels << std::endl;

	hipdnnTensorDescriptor_t output_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
											/*format=*/HIPDNN_TENSOR_NHWC,
											/*dataType=*/HIPDNN_DATA_FLOAT,
											/*batch_size=*/batch_size,
											/*channels=*/channels,
											/*image_height=*/height,
											/*image_width=*/width));

	hipdnnConvolutionFwdAlgo_t convolution_algorithm;
	checkCUDNN(
		hipdnnGetConvolutionForwardAlgorithm(gCudnnHandle,
											input_descriptor,
											kernel_descriptor,
											convolution_descriptor,
											output_descriptor,
											HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
											/*memoryLimitInBytes=*/0,
											&convolution_algorithm));

	size_t workspace_bytes{0};
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(gCudnnHandle,
														input_descriptor,
														kernel_descriptor,
														convolution_descriptor,
														output_descriptor,
														convolution_algorithm,
														&workspace_bytes));
	std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
				<< std::endl;
	// assert(workspace_bytes > 0);

	void* d_workspace{nullptr};
	gpuErrChk(hipMalloc(&d_workspace, workspace_bytes));

	int image_bytes = batch_size * channels * height * width * sizeof(float);
	assert(image_bytes == ip_width * ip_height * 4*sizeof(float));

	float* d_input{nullptr};
	gpuErrChk(hipMalloc(&d_input, ip_height * ip_width * 4 * sizeof(float)));

	ILubyte* bytes = ilGetData();
	float *bytes2 = new float[ip_height * ip_width * 4];
	for (int i = 0; i < ip_height * ip_width * 4; i++)
	{
		bytes2[i] = bytes[i] / 255.0;
	}
	printf("copied image to input\n");

	gpuErrChk(hipMemcpy(d_input, bytes2, ip_height * ip_width * 4 * sizeof(float), hipMemcpyHostToDevice));

	printf("copied input to GPU\n");

	float* d_output{nullptr};
	hipMalloc(&d_output, image_bytes);
	hipMemset(d_output, 0, image_bytes);

	// clang-format off
	const float kernel_template[3][3] = {
		{1, 1, 1},
		{1, -8, 1},
		{1, 1, 1}
	};
	// clang-format on

	float h_kernel[4][4][3][3];
	for (int kernel = 0; kernel < 4; ++kernel) {
		for (int channel = 0; channel < 4; ++channel) {
		for (int row = 0; row < 3; ++row) {
			for (int column = 0; column < 3; ++column) {
			h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		}
		}
	}

	float* d_kernel{nullptr};
	hipMalloc(&d_kernel, sizeof(h_kernel));
	hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

	const float alpha = 1.0f, beta = 0.0f;

	checkCUDNN(hipdnnConvolutionForward(gCudnnHandle,
										&alpha,
										input_descriptor,
										d_input,
										kernel_descriptor,
										d_kernel,
										convolution_descriptor,
										convolution_algorithm,
										d_workspace,
										workspace_bytes,
										&beta,
										output_descriptor,
										d_output));

	// if (with_sigmoid) {
	// 	hipdnnActivationDescriptor_t activation_descriptor;
	// 	checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
	// 	checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
	// 											HIPDNN_ACTIVATION_SIGMOID,
	// 											HIPDNN_PROPAGATE_NAN,
	// 											/*relu_coef=*/0));
	// 	checkCUDNN(hipdnnActivationForward(cudnn,
	// 									activation_descriptor,
	// 									&alpha,
	// 									output_descriptor,
	// 									d_output,
	// 									&beta,
	// 									output_descriptor,
	// 									d_output));
	// 	hipdnnDestroyActivationDescriptor(activation_descriptor);
	// }

	float* h_output = new float[image_bytes];
	hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < image_bytes;i++)
	{
		if(h_output[i]!=0)
			printf("%f ", h_output[i]);
	}

	ILuint imageName2;
	ilGenImages(1, &imageName2);
	ilBindImage(imageName2);
	ILubyte *bytes3 = new ILubyte[width * height * 4];
	ilTexImage(width, height, 1, 4, IL_RGBA, IL_UNSIGNED_BYTE, bytes3);
	// ilSetInteger(IL_IMAGE_WIDTH, width);
	// ilSetInteger(IL_IMAGE_HEIGHT, height);
	printf("copying output\n");
	for (int i = 0; i < ip_height * ip_width * 4; i++)
	{
		bytes3[i] = std::max(0.0,std::min(255.0,h_output[i] * 255.0));
	}
	auto error = ilGetError();
	printf("error %d\n",error);
	ilSetPixels(0, 0, 0, ip_width, ip_height, 4, IL_RGBA, IL_UNSIGNED_BYTE, bytes3);
	// ilSetInteger(IL_IMAGE_WIDTH, width);
	// ilSetInteger(IL_IMAGE_HEIGHT, height);
	error = ilGetError();
	printf("error %d\n",error);
	// printf(iluErrorString(error));
	printf("saving\n");
	ilEnable(IL_FILE_OVERWRITE);
	ilSaveImage("output2.png");
	error = ilGetError();
	printf("error %d\n",error);
}


void test_cudnn_conv()
{
	ilInit();
	std::string path = "tensorflow.png";
	// load_image("tensorflow.png");
	ILuint imageName;
	ilGenImages(1, &imageName);
	ilBindImage(imageName);
	ilLoadImage(path.c_str());
	// auto view = boost::gil::view(img);

	// auto channels = view.num_channels();
	// auto dim = view.dimensions();
	// printf("dim %d %d %d %d\n", dim.x, dim.y, dim.num_dimensions, channels);

	int width = ilGetInteger(IL_IMAGE_WIDTH);
	int height = ilGetInteger(IL_IMAGE_HEIGHT);
	int channels = ilGetInteger(IL_IMAGE_CHANNELS);

	printf("dim %d %d %d\n", width, height, channels);

	Blob *input = new Blob(make_shape(1,4,height,width)); 
	printf("bdim %d %d %d\n", width, height, channels);
	Blob *output = new Blob(make_shape(1,4,height,width));
	printf("adim %d %d %d\n", width, height, channels);
	// output->Data.setconstant(255);
	// output->Data.copyToGPU();
	ConvNeuron *neuron = new ConvNeuron(input, output, 3, 3, 1, 1, 1, 1);

	ILubyte* bytes = ilGetData();
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			// printf( "%s\n", "Red Value for Pixel");
			// printf( "%d ", bytes[(i*width + j)*4 + 0]);
			input->Data(0, 0, i, j) = bytes[(i * width + j) * 4 + 0]/255.0;
			// printf("%s\n", "Green Value for Pixel");
			// printf( "%d\n", bytes[(i*width + j)*4 + 1]);
			input->Data(0, 1, i, j) = bytes[(i * width + j) * 4 + 1]/255.0;
			// printf( "%s\n", "Blue Value for Pixel");
			// printf( "%d\n", bytes[(i*width + j)*4 + 2]);
			input->Data(0, 2, i, j) = bytes[(i * width + j) * 4 + 2]/255.0;
			input->Data(0, 3, i, j) = bytes[(i * width + j) * 4 + 3]/255.0;
			// input->Data(0, 3, j, i) = bytes[(i * height + j) * 4 + 3] = 1.0f;
		}
	}
	// for (int i = 0;i<height*width*4;i++)
	// {
	// 	input->Data(i) = bytes[i] / 255.0;
	// }
	printf("copied image to input\n");
	input->Data.copyToGPU();

	// Mystery kernel
	const float kernel_template[3][3] = {
	{1,  1, 1},
	{1, -8, 1},
	{1,  1, 1}
	};

	for (int i = 0; i < 4;i++)
	{
		printf("dim %d %d, ", i, neuron->Weights->Data.mAllocShape[i]);
		assert(neuron->Weights->Data.mAllocShape[i] == neuron->Weights->Data.mShape[i]);
	}

	// float h_kernel[3][3][3][3];
	for (int kernel = 0; kernel < 4; ++kernel)
	{
		for (int channel = 0; channel < 4; ++channel)
		{
			for (int row = 0; row < 3; ++row)
			{
				for (int column = 0; column < 3; ++column)
				{
					neuron->Weights->Data(kernel, channel, row, column) = kernel_template[row][column];
					// neuron->Weights->Data(kernel,channel,row,column) = (rand()%1024)/1024;
				}
			}
		}
	}
	printf("set kernel\n");
	neuron->Weights->Data.copyToGPU();

	neuron->forward();
	output->Data.copyToCPU();

	ILuint imageName2;
	ilGenImages(1, &imageName2);
	ilBindImage(imageName2);
	// ilSetInteger(IL_IMAGE_WIDTH, width);
	// ilSetInteger(IL_IMAGE_HEIGHT, height);
	ILubyte *bytes2 = new ILubyte[width * height * 4];
	ilTexImage(width, height, 1, 4, IL_RGBA, IL_UNSIGNED_BYTE, bytes2);

	printf("copying output\n");
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			// printf( "%s\n", "Red Value for Pixel");
			// printf( "%d\n", bytes[(i*width + j)*4 + 0]);
			bytes2[(i * width + j) * 4 + 0] = std::max(0.0f,std::min(255.0f,output->Data(0,0,i,j)*255.0f));
			bytes2[(i * width + j) * 4 + 1] = std::max(0.0f,std::min(255.0f,output->Data(0,1,i,j)*255.0f));
			bytes2[(i * width + j) * 4 + 2] = std::max(0.0f,std::min(255.0f,output->Data(0,2,i,j)*255.0f));
			// bytes2[(i * height + j) * 4 + 3] = std::max(0.0f,std::min(255.0f,output->Data(0,3,j,i)*255.0f));
			bytes2[(i * width + j) * 4 + 3] = 255;
			// bytes2[(i * height + j) * 4 + 0] = 255;
			// printf("%f %f %f ", output->Data(0, 0, j, i), output->Data(0, 1, j, i), output->Data(0, 2, j, i));
			// printf("%s\n", "Green Value for Pixel");
			// printf( "%d\n", bytes[(i*width + j)*4 + 1]);
			// input->Data(0, 1, j, i) = bytes[(i * width + j) * 4 + 1];
			// printf( "%s\n", "Blue Value for Pixel");
			// printf( "%d\n", bytes[(i*width + j)*4 + 2]);
			// input->Data(0, 2, j, i) = bytes[(i * width + j) * 4 + 2];
		}
	}
	// for (int i = 0; i < height * width * 4;i++)
	// {
	// 	bytes2[i] = std::max(0.0f,std::min(255.0f,output->Data(i)*255.0f));
	// 	// printf("%f ", output->Data(i));
	// 	// if(i%4==3)
	// 	// {
	// 	// 	bytes2[i] = 255;
	// 	// }
	// }
	auto error = ilGetError();
	printf("error %d\n",error);
	ilSetPixels(0, 0, 0, width, height, 1, IL_RGBA, IL_UNSIGNED_BYTE, bytes2);
	error = ilGetError();
	printf("error %d\n",error);
	// printf(iluErrorString(error));
	printf("saving\n");
	ilEnable(IL_FILE_OVERWRITE);
	ilSaveImage("output.png");
	error = ilGetError();
	printf("error %d\n",error);
	// printf("%s\n", iluErrorString(error));
	// printf("Output in output.png\n");
	// save_image("cudnn-out.png", output->Data.mData, output->Data.mShape[2], output->Data.mShape[3]);
	//extract output
}