#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
// #include <>

#include "CudaTests.h"

void test_cublas_vector_add()
{
    int n = 1 << 5;
    size_t bytes = n * sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b;

    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_c = (float *)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);

    for (int i = 0; i < n;i++)
    {
        h_a[i] = i+1;
        h_b[i] = 2 * i+1;
    }

    // hipblasHandle_t handle;
    // hipblasCreate(&handle);

    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

    const float scale = 2.0f;
    hipblasSaxpy(gCublasHandle, n, &scale, d_a, 1, d_b, 1);
    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

    for (int i = 0; i < n;i++)
    {
        printf("%f\n", h_c[i]);
    }

    // hipblasDestroy(gCublasHandle);

    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);
}

// Verify our result on the CPU
// Indexing must account for the CUBLAS operating on column-major data
void verify_solution(float *a, float *b, float *c, int M, int N, int K) {
  // Tolerance for our result (floats are imperfect)
  float epsilon = 0.001f;

  // For every row...
  for (int row = 0; row < M; row++) {
    // For every column
    for (int col = 0; col < N; col++) {
      // For every element in the row-col pair...
      float temp = 0;
      for (int i = 0; i < K; i++) {
        temp += a[row + M * i] * b[col * K + i];
      }

      // Check to see if the difference falls within our tolerance
      assert(fabs(c[col * M + row] - temp) <= epsilon);
    }
  }
}

void cublas_matmul()
{
  // Dimensions for our matrices
  // MxK * KxN = MxN
  const int M = 1 << 9;
  const int N = 1 << 8;
  const int K = 1 << 7;

  // Pre-calculate the size (in bytes) of our matrices
  const size_t bytes_a = M * K * sizeof(float);
  const size_t bytes_b = K * N * sizeof(float);
  const size_t bytes_c = M * N * sizeof(float);

  // Vectors for the host data
  std::vector<float> h_a(M * K);
  std::vector<float> h_b(K * N);
  std::vector<float> h_c(M * N);
  
  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // Pseudo random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set the seed
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  // Fill the matrix with random numbers on the device
  hiprandGenerateUniform(prng, d_a, M * K);
  hiprandGenerateUniform(prng, d_b, K * M);

  // cuBLAS handle
  // hipblasHandle_t handle;
  // hipblasCreate(&handle);

  // Scalaing factors
  float alpha = 1.0f;
  float beta = 0.0f;

  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb,
  // beta, C, ldc
  hipblasSgemm(gCublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, M, d_b, K,
              &beta, d_c, M);

  // Copy back the three matrices
  hipMemcpy(h_a.data(), d_a, bytes_a, hipMemcpyDeviceToHost);
  hipMemcpy(h_b.data(), d_b, bytes_b, hipMemcpyDeviceToHost);
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);

  // Verify solution
  verify_solution(h_a.data(), h_b.data(), h_c.data(), M, N, K);
  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free our memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

void test_cugemm()
{
	Tensor t1(make_shape(2, 3));
	t1(0, 0) = -1;
	t1(0, 1) = 1;
	t1(0, 2) = 4;
	t1(1, 0) = -4;
	t1(1, 1) = 0;
	t1(1, 2) = -3;
	t1.print();

	Tensor t1_t(make_shape(3, 2));
	t1_t(0, 0) = -1;
	t1_t(1, 0) = 1;
	t1_t(2, 0) = 4;
	t1_t(0, 1) = -4;
	t1_t(1, 1) = 0;
	t1_t(2, 1) = -3;
	t1_t.print();

	Tensor t2(make_shape(4, 2));
	t2(0, 0) = 2;
	t2(1, 0) = 3;
	t2(2, 0) = -2;
	t2(3, 0) = 1;
	t2(0, 1) = 4;
	t2(1, 1) = 0;
	t2(2, 1) = 5;
	t2(3, 1) = 6;
	t2.print();

	Tensor t2_t(make_shape(2, 4));
	t2_t(0, 0) = 2;
	t2_t(0, 1) = 3;
	t2_t(0, 2) = -2;
	t2_t(0, 3) = 1;
	t2_t(1, 0) = 4;
	t2_t(1, 1) = 0;
	t2_t(1, 2) = 5;
	t2_t(1, 3) = 6;
	t2_t.print();

	t1.copyToGPU();
	t1_t.copyToGPU();
	t2.copyToGPU();
	t2_t.copyToGPU();

	// printf("GPU print t1\n");
	// printGPU<<<1, 1>>>(t1.mShape[0], t1.mShape[1], t1.mLD, t1.mDataGPU);

	Tensor t3(make_shape(4, 3));

	//Mat Mul
	/*clblasDgemm(clblasRowMajor, clblasNoTrans, clblasNoTrans, t1.cols(), t2.rows(),
	t1.rows(), 1, t1.mData, t1.rows(), t2.mData, t2.rows(), 0, t3.mData, t3.rows())*/
	//cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, t1.rows(), t2.cols(),
	//	t1.cols(), 1, t1.mData, t1.cols(), t2.mData, t2.cols(), 0, t3.mData, t3.cols());
	gemm_gpu(&t1, &t2, &t3, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 0);
	// hipDeviceSynchronize();
	printf("1\n");
	// printGPU<<<1, 1>>>(t3.mShape[0],t3.mShape[1],t3.mLD,t3.mDataGPU);
	// hipDeviceSynchronize();
	t3.copyToCPU();
	t3.print();
	gemm_gpu(&t1_t, &t2_t, &t3, HIPBLAS_OP_T, HIPBLAS_OP_T, 1, 0);
	t3.copyToCPU();
	printf("2\n");
	t3.print();
	gemm_gpu(&t1, &t2_t, &t3, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, 0);
	t3.copyToCPU();
	printf("3\n");
	t3.print();
	gemm_gpu(&t1_t, &t2, &t3, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, 0);
	t3.copyToCPU();
	printf("4\n");
	t3.print();
	
	// Result should be:
	//  30  29  43  45
	// -29 -36 -19 -34

	printf("freeing mem\n");
	t1.freemem();
	t1_t.freemem();
	t2.freemem();
	t2_t.freemem();
	t3.freemem();
}

void test_cugemm_symm()
{
	Tensor t1(make_shape(2, 2));
	t1(0, 0) = -1;
	t1(0, 1) = 1;
	t1(1, 0) = -4;
	t1(1, 1) = 0;
	t1.print();

	Tensor t2(make_shape(2, 2));
	t2(0, 0) = 2;
	t2(0, 1) = 3;
	t2(1, 0) = 4;
	t2(1, 1) = 0;
	t2.print();

	t1.copyToGPU();
	// t1_t.copyToGPU();
	t2.copyToGPU();
	// t2_t.copyToGPU();

	printf("GPU print t1\n");
	printGPU<<<1, 1>>>(t1.mShape[0], t1.mShape[1], t1.mLD, t1.mDataGPU);

	Tensor t3(make_shape(2, 2));

	//Mat Mul
	/*clblasDgemm(clblasRowMajor, clblasNoTrans, clblasNoTrans, t1.cols(), t2.rows(),
	t1.rows(), 1, t1.mData, t1.rows(), t2.mData, t2.rows(), 0, t3.mData, t3.rows())*/
	//cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, t1.rows(), t2.cols(),
	//	t1.cols(), 1, t1.mData, t1.cols(), t2.mData, t2.cols(), 0, t3.mData, t3.cols());
	gemm_gpu(&t1, &t2, &t3, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 0);
	// hipDeviceSynchronize();
	printf("1\n");
	// printGPU<<<1, 1>>>(t3.mShape[0],t3.mShape[1],t3.mLD,t3.mDataGPU);
	// hipDeviceSynchronize();
	t3.copyToCPU();
	t3.print();
	// gemm_gpu(&t1_t, &t2_t, &t3, HIPBLAS_OP_T, HIPBLAS_OP_T, 1, 0);
	// t3.copyToCPU();
	// printf("2\n");
	// t3.print();
	// gemm_gpu(&t1, &t2_t, &t3, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, 0);
	// t3.copyToCPU();
	// printf("3\n");
	// t3.print();
	// gemm_gpu(&t1_t, &t2, &t3, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, 0);
	// t3.copyToCPU();
	// printf("4\n");
	// t3.print();
	
	// Result should be:
	//  30  29  43  45
	// -29 -36 -19 -34

	printf("freeing mem\n");
	t1.freemem();
	// t1_t.freemem();
	t2.freemem();
	// t2_t.freemem();
	t3.freemem();
}