#include "hip/hip_runtime.h"
#include "Tensor.h"

void Tensor::allocateGPU()
{
	// if (mMemory != NULL)
	// {
	// 	freeGPU();
	// }
	// cl_int err;
	// mMemory = clCreateBuffer(gCLContext, CL_MEM_READ_WRITE, mSize * sizeof(cl_float), NULL, &err);
	// if (err != CL_SUCCESS)
	// {
	// 	printf("ERROR: allocating tensor GPU: %d\n", err);
	// }
#ifdef USE_GPU
	if(mDataGPU!=NULL)
	{
		freeGPU();
	}
	gpuErrChk(hipMalloc(&mDataGPU, mAllocSize * sizeof(Float)));
#endif
}


void Tensor::freeGPU()
{
	// if (mMemory != NULL)
	// {
	// 	clReleaseMemObject(mMemory);
	// 	mMemory = NULL;
	// }
#ifdef USE_GPU
	if(mDataGPU!=NULL)
	{
		gpuErrChk(hipFree(mDataGPU));
		mDataGPU = NULL;
	}
#endif
}

void Tensor::copyToGPU()
{
#ifdef USE_GPU
	gpuErrChk(hipMemcpy(mDataGPU, mData, mAllocSize * sizeof(Float), hipMemcpyHostToDevice));
#endif
	// if (mMemory != NULL && mData != NULL)
	// {
	// 	cl_int err = clEnqueueWriteBuffer(gCLQueue, mMemory, CL_TRUE, 0, mSize * sizeof(cl_float), mData, 0, NULL, NULL);
	// 	if (err != CL_SUCCESS)
	// 	{
	// 		printf("ERROR: copytoGPU: %d\n", err);
	// 	}
	// }



	// /*cl_int err = clblasWriteMatrix(clblasRowMajor, mSize * sizeof(cl_float), mSize * sizeof(cl_float), sizeof(cl_float),
	// mData, 0, cols(), mMemory, 0, cols(),
	// gCLQueue, 1, NULL);
	// if (err != CL_SUCCESS)
	// {
	// printf("ERROR: copytoGPU: %d\n", err);
	// }*/
}

void Tensor::copyToCPU()
{
#ifdef USE_GPU
	gpuErrChk(hipMemcpy(mData, mDataGPU, mAllocSize * sizeof(Float), hipMemcpyDeviceToHost));
#endif
	// if (mMemory != NULL && mData != NULL)
	// {
	// 	cl_int err = clEnqueueReadBuffer(gCLQueue, mMemory, CL_TRUE, 0, mSize * sizeof(cl_float), mData, 0, NULL, NULL);
	// 	if (err != CL_SUCCESS)
	// 	{
	// 		printf("ERROR: copytoCPU: %d\n", err);
	// 	}
	// }
	// /*cl_int err = clblasReadMatrix(clblasRowMajor, mSize * sizeof(cl_float), mSize * sizeof(cl_float), sizeof(cl_float),
	// mMemory, 0, cols(), mData, 0, cols(),
	// gCLQueue, 1, NULL);
	// if (err != CL_SUCCESS)
	// {
	// printf("ERROR: copytoGPU: %d\n", err);
	// }*/
}

// __global__ void Tensor::printGPU() const
// {
// 	for (uint64_t i = 0; i < mShape[0]; i++)
// 	{
// 		for (uint64_t j = 0; j < mShape[1]; j++)
// 		{
// 			printf("%f ", mDataGPU[i*mLD+j]);
// 		}
// 		printf("\n");
// 	}
// }


__global__ void printGPU(int m, int n, int ld, float* data)
{
	for (uint64_t i = 0; i < n; i++)
	{
		for (uint64_t j = 0; j < m; j++)
		{
			printf("%f ", data[j*ld+i]);
		}
		printf("\n");
	}
}