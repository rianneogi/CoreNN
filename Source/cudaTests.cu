#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
// #include <>
#include <hipblas.h>
#include <stdlib.h>
#include <stdio.h>
#include <cstdio>
#include <hiprand.h>
#include <cassert>
#include <cmath>
#include <ctime>
#include <iostream>
#include <vector>

#include "cudaTests.h"

void cublas_vector_add()
{
    int n = 1 << 5;
    size_t bytes = n * sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b;

    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_c = (float *)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);

    for (int i = 0; i < n;i++)
    {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

    const float scale = 2.0f;
    hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1);
    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

    for (int i = 0; i < n;i++)
    {
        printf("%f\n", h_c[i]);
    }

    hipblasDestroy(handle);

    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);
}

// Verify our result on the CPU
// Indexing must account for the CUBLAS operating on column-major data
void verify_solution(float *a, float *b, float *c, int M, int N, int K) {
  // Tolerance for our result (floats are imperfect)
  float epsilon = 0.001f;

  // For every row...
  for (int row = 0; row < M; row++) {
    // For every column
    for (int col = 0; col < N; col++) {
      // For every element in the row-col pair...
      float temp = 0;
      for (int i = 0; i < K; i++) {
        temp += a[row + M * i] * b[col * K + i];
      }

      // Check to see if the difference falls within our tolerance
      assert(fabs(c[col * M + row] - temp) <= epsilon);
    }
  }
}

void cublas_matmul()
{
  // Dimensions for our matrices
  // MxK * KxN = MxN
  const int M = 1 << 9;
  const int N = 1 << 8;
  const int K = 1 << 7;

  // Pre-calculate the size (in bytes) of our matrices
  const size_t bytes_a = M * K * sizeof(float);
  const size_t bytes_b = K * N * sizeof(float);
  const size_t bytes_c = M * N * sizeof(float);

  // Vectors for the host data
  std::vector<float> h_a(M * K);
  std::vector<float> h_b(K * N);
  std::vector<float> h_c(M * N);
  
  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // Pseudo random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set the seed
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  // Fill the matrix with random numbers on the device
  hiprandGenerateUniform(prng, d_a, M * K);
  hiprandGenerateUniform(prng, d_b, K * M);

  // cuBLAS handle
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Scalaing factors
  float alpha = 1.0f;
  float beta = 0.0f;

  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb,
  // beta, C, ldc
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, M, d_b, K,
              &beta, d_c, M);

  // Copy back the three matrices
  hipMemcpy(h_a.data(), d_a, bytes_a, hipMemcpyDeviceToHost);
  hipMemcpy(h_b.data(), d_b, bytes_b, hipMemcpyDeviceToHost);
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);

  // Verify solution
  verify_solution(h_a.data(), h_b.data(), h_c.data(), M, N, K);
  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free our memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}